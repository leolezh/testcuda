  __global__  void vecAdd(float* A, float* B, float* C)
       {
          // threadIdx.x is a built-in variable  provided by CUDA at runtime
          int i = threadIdx.x;
          A[i]=0;
          B[i]=i;
          C[i] = A[i] + B[i];
          //printf("from thread No.%d",threadIdx.x);
       }

       
#include <hip/hip_runtime.h>
#include  <stdio.h>
       #define  SIZE 10
       int  main()
       {
        int N=SIZE;
        float A[SIZE], B[SIZE], C[SIZE];
        float *devPtrA;
        float *devPtrB;
        float *devPtrC;
        int memsize= SIZE * sizeof(float);
        hipMalloc(&devPtrA, memsize);
        hipMalloc(&devPtrB, memsize);
        hipMalloc((void**)&devPtrC, memsize); // the same as cudaMalloc(&devPtrC, memsize);


        hipMemcpy(devPtrA, A, memsize,  hipMemcpyHostToDevice);
        hipMemcpy(devPtrB, B, memsize,  hipMemcpyHostToDevice);
        // __global__ functions are called:  Func<<< Dg, Db, Ns  >>>(parameter);
        vecAdd<<<1, N>>>(devPtrA,  devPtrB, devPtrC);
        hipError_t error = hipGetLastError();
        if(error != hipSuccess)
        {
                printf("CUDA error: errorstring %s\n", hipGetErrorString(error));
                exit(-1);
        }

        hipMemcpy(C, devPtrC, memsize,  hipMemcpyDeviceToHost);

        for (int i=0; i<SIZE; i++)
           printf("C[%d]=%f\n",i,C[i]);

        hipFree(devPtrA);
        hipFree(devPtrA);
        hipFree(devPtrA);
       }
